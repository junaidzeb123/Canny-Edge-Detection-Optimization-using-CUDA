/*******************************************************************************
 * FILE: hysteresis.c
 * This code was re-written by Mike Heath from original code obtained indirectly
 * from Michigan State University. heath@csee.usf.edu (Re-written in 1996).
 *******************************************************************************/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

#define VERBOSE 0

#define NOEDGE 255
#define POSSIBLE_EDGE 128
#define EDGE 0

/*******************************************************************************
 * PROCEDURE: follow_edges
 * PURPOSE: This procedure edges is a recursive routine that traces edgs along
 * all paths whose magnitude values remain above some specifyable lower
 * threshhold.
 * NAME: Mike Heath
 * DATE: 2/15/96
 *******************************************************************************/
static int times = 0;
void follow_edges(unsigned char *edgemapptr, short *edgemagptr, short lowval,
                  int cols) {
    times++;
    short *tempmagptr;
    unsigned char *tempmapptr;
    int i;
    float thethresh;
    int x[8] = {1, 1, 0, -1, -1, -1, 0, 1},
        y[8] = {0, 1, 1, 1, 0, -1, -1, -1};

    for (i = 0; i < 8; i++) {
        tempmapptr = edgemapptr - y[i] * cols + x[i];
        tempmagptr = edgemagptr - y[i] * cols + x[i];

        if ((*tempmapptr == POSSIBLE_EDGE) && (*tempmagptr > lowval)) {
            *tempmapptr = (unsigned char)EDGE;
            follow_edges(tempmapptr, tempmagptr, lowval, cols);
        }
    }
}

/*******************************************************************************
 * PROCEDURE: apply_hysteresis
 * PURPOSE: This routine finds edges that are above some high threshhold or
 * are connected to a high pixel by a path of pixels greater than a low
 * threshold.
 * NAME: Mike Heath
 * DATE: 2/15/96
 *******************************************************************************/
void apply_hysteresis(short int *mag, unsigned char *nms, int rows, int cols,
                      float tlow, float thigh, unsigned char *edge) {
    int r, c, pos, numedges, lowcount, highcount, lowthreshold, highthreshold,
        i, hist[32768], rr, cc;
    short int maximum_mag, sumpix;

    /****************************************************************************
     * Initialize the edge map to possible edges everywhere the non-maximal
     * suppression suggested there could be an edge except for the border. At
     * the border we say there can not be an edge because it makes the
     * follow_edges algorithm more efficient to not worry about tracking an
     * edge off the side of the image.
     ****************************************************************************/
    for (r = 0, pos = 0; r < rows; r++) {
        for (c = 0; c < cols; c++, pos++) {
            if (nms[pos] == POSSIBLE_EDGE)
                edge[pos] = POSSIBLE_EDGE;
            else
                edge[pos] = NOEDGE;
        }
    }

    for (r = 0, pos = 0; r < rows; r++, pos += cols) {
        edge[pos] = NOEDGE;
        edge[pos + cols - 1] = NOEDGE;
    }
    pos = (rows - 1) * cols;
    for (c = 0; c < cols; c++, pos++) {
        edge[c] = NOEDGE;
        edge[pos] = NOEDGE;
    }

    /****************************************************************************
     * Compute the histogram of the magnitude image. Then use the histogram to
     * compute hysteresis thresholds.
     ****************************************************************************/
    for (r = 0; r < 32768; r++)
        hist[r] = 0;
    for (r = 0, pos = 0; r < rows; r++) {
        for (c = 0; c < cols; c++, pos++) {
            if (edge[pos] == POSSIBLE_EDGE)
                hist[mag[pos]]++;
        }
    }

    /****************************************************************************
     * Compute the number of pixels that passed the nonmaximal suppression.
     ****************************************************************************/
    for (r = 1, numedges = 0; r < 32768; r++) {
        if (hist[r] != 0)
            maximum_mag = r;
        numedges += hist[r];
    }

    highcount = (int)(numedges * thigh + 0.5);

    /****************************************************************************
     * Compute the high threshold value as the (100 * thigh) percentage point
     * in the magnitude of the gradient histogram of all the pixels that passes
     * non-maximal suppression. Then calculate the low threshold as a fraction
     * of the computed high threshold value. John Canny said in his paper
     * "A Computational Approach to Edge Detection" that "The ratio of the
     * high to low threshold in the implementation is in the range two or three
     * to one." That means that in terms of this implementation, we should
     * choose tlow ~= 0.5 or 0.33333.
     ****************************************************************************/
    r = 1;
    numedges = hist[1];
    while ((r < (maximum_mag - 1)) && (numedges < highcount)) {
        r++;
        numedges += hist[r];
    }
    highthreshold = r;
    lowthreshold = (int)(highthreshold * tlow + 0.5);

    if (VERBOSE) {
        printf("The input low and high fractions of %f and %f computed to\n",
               tlow, thigh);
        printf("magnitude of the gradient threshold values of: %d %d\n",
               lowthreshold, highthreshold);
    }

    /****************************************************************************
     * This loop looks for pixels above the highthreshold to locate edges and
     * then calls follow_edges to continue the edge.
     ****************************************************************************/
    for (r = 0, pos = 0; r < rows; r++) {
        for (c = 0; c < cols; c++, pos++) {
            if ((edge[pos] == POSSIBLE_EDGE) && (mag[pos] >= highthreshold)) {
                edge[pos] = EDGE;
                follow_edges((edge + pos), (mag + pos), lowthreshold, cols);
            }
        }
    }
    printf("%d\n", times);

    /****************************************************************************
     * Set all the remaining possible edges to non-edges.
     ****************************************************************************/
    for (r = 0, pos = 0; r < rows; r++) {
        for (c = 0; c < cols; c++, pos++)
            if (edge[pos] != EDGE)
                edge[pos] = NOEDGE;
    }
}

/*******************************************************************************
 * PROCEDURE: non_max_supp
 * PURPOSE: This routine applies non-maximal suppression to the magnitude of
 * the gradient image.
 * NAME: Mike Heath
 * DATE: 2/15/96
 *******************************************************************************/
void non_max_supp(short *mag, short *gradx, short *grady, int nrows, int ncols,
                  unsigned char *result) {
    int rowcount, colcount, count;
    short *magrowptr, *magptr;
    short *gxrowptr, *gxptr;
    short *gyrowptr, *gyptr, z1, z2;
    short m00, gx, gy;
    float mag1, mag2, xperp, yperp;
    unsigned char *resultrowptr, *resultptr;

    /****************************************************************************
     * Zero the edges of the result image.
     ****************************************************************************/
    for (count = 0, resultrowptr = result, resultptr = result + ncols * (nrows - 1);
         count < ncols; resultptr++, resultrowptr++, count++) {
        *resultrowptr = *resultptr = (unsigned char)0;
    }

    for (count = 0, resultptr = result, resultrowptr = result + ncols - 1;
         count < nrows; count++, resultptr += ncols, resultrowptr += ncols) {
        *resultptr = *resultrowptr = (unsigned char)0;
    }

    /****************************************************************************
     * Suppress non-maximum points.
     ****************************************************************************/
    for (rowcount = 1, magrowptr = mag + ncols + 1, gxrowptr = gradx + ncols + 1,
        gyrowptr = grady + ncols + 1, resultrowptr = result + ncols + 1;
         rowcount < nrows - 2;
         rowcount++, magrowptr += ncols, gyrowptr += ncols, gxrowptr += ncols,
        resultrowptr += ncols) {
        for (colcount = 1, magptr = magrowptr, gxptr = gxrowptr, gyptr = gyrowptr,
            resultptr = resultrowptr;
             colcount < ncols - 2;
             colcount++, magptr++, gxptr++, gyptr++, resultptr++) {
            m00 = *magptr;
            if (m00 == 0) {
                *resultptr = (unsigned char)NOEDGE;
            } else {
                xperp = -(gx = *gxptr) / ((float)m00);
                yperp = (gy = *gyptr) / ((float)m00);
            }

            if (gx >= 0) {
                if (gy >= 0) {
                    if (gx >= gy) {
                        /* 111 */
                        /* Left point */
                        z1 = *(magptr - 1);
                        z2 = *(magptr - ncols - 1);

                        mag1 = (m00 - z1) * xperp + (z2 - z1) * yperp;

                        /* Right point */
                        z1 = *(magptr + 1);
                        z2 = *(magptr + ncols + 1);

                        mag2 = (m00 - z1) * xperp + (z2 - z1) * yperp;
                    } else {
                        /* 110 */
                        /* Left point */
                        z1 = *(magptr - ncols);
                        z2 = *(magptr - ncols - 1);

                        mag1 = (z1 - z2) * xperp + (z1 - m00) * yperp;

                        /* Right point */
                        z1 = *(magptr + ncols);
                        z2 = *(magptr + ncols + 1);

                        mag2 = (z1 - z2) * xperp + (z1 - m00) * yperp;
                    }
                } else {
                    if (gx >= -gy) {
                        /* 101 */
                        /* Left point */
                        z1 = *(magptr - 1);
                        z2 = *(magptr + ncols - 1);

                        mag1 = (m00 - z1) * xperp + (z1 - z2) * yperp;

                        /* Right point */
                        z1 = *(magptr + 1);
                        z2 = *(magptr - ncols + 1);

                        mag2 = (m00 - z1) * xperp + (z1 - z2) * yperp;
                    } else {
                        /* 100 */
                        /* Left point */
                        z1 = *(magptr + ncols);
                        z2 = *(magptr + ncols - 1);

                        mag1 = (z1 - z2) * xperp + (m00 - z1) * yperp;

                        /* Right point */
                        z1 = *(magptr - ncols);
                        z2 = *(magptr - ncols + 1);

                        mag2 = (z1 - z2) * xperp + (m00 - z1) * yperp;
                    }
                }
            } else {
                if ((gy = *gyptr) >= 0) {
                    if (-gx >= gy) {
                        /* 011 */
                        /* Left point */
                        z1 = *(magptr + 1);
                        z2 = *(magptr - ncols + 1);

                        mag1 = (z1 - m00) * xperp + (z2 - z1) * yperp;

                        /* Right point */
                        z1 = *(magptr - 1);
                        z2 = *(magptr + ncols - 1);

                        mag2 = (z1 - m00) * xperp + (z2 - z1) * yperp;
                    } else {
                        /* 010 */
                        /* Left point */
                        z1 = *(magptr - ncols);
                        z2 = *(magptr - ncols + 1);

                        mag1 = (z2 - z1) * xperp + (z1 - m00) * yperp;

                        /* Right point */
                        z1 = *(magptr + ncols);
                        z2 = *(magptr + ncols - 1);

                        mag2 = (z2 - z1) * xperp + (z1 - m00) * yperp;
                    }
                } else {
                    if (-gx > -gy) {
                        /* 001 */
                        /* Left point */
                        z1 = *(magptr + 1);
                        z2 = *(magptr + ncols + 1);

                        mag1 = (z1 - m00) * xperp + (z1 - z2) * yperp;

                        /* Right point */
                        z1 = *(magptr - 1);
                        z2 = *(magptr - ncols - 1);

                        mag2 = (z1 - m00) * xperp + (z1 - z2) * yperp;
                    } else {
                        /* 000 */
                        /* Left point */
                        z1 = *(magptr + ncols);
                        z2 = *(magptr + ncols + 1);

                        mag1 = (z2 - z1) * xperp + (m00 - z1) * yperp;

                        /* Right point */
                        z1 = *(magptr - ncols);
                        z2 = *(magptr - ncols - 1);

                        mag2 = (z2 - z1) * xperp + (m00 - z1) * yperp;
                    }
                }
            }

            /* Now determine if the current point is a maximum point */

            if ((mag1 > 0.0) || (mag2 > 0.0)) {
                *resultptr = (unsigned char)NOEDGE;
            } else {
                if (mag2 == 0.0)
                    *resultptr = (unsigned char)NOEDGE;
                else
                    *resultptr = (unsigned char)POSSIBLE_EDGE;
            }
        }
    }
}

/*******************************************************************************
 * FILE: pgm_io.c
 * This code was written by Mike Heath. heath@csee.usf.edu (in 1995).
 *******************************************************************************/

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

/******************************************************************************
 * Function: read_pgm_image
 * Purpose: This function reads in an image in PGM format. The image can be
 * read in from either a file or from standard input. The image is only read
 * from standard input when infilename = NULL. Because the PGM format includes
 * the number of columns and the number of rows in the image, these are read
 * from the file. Memory to store the image is allocated in this function.
 * All comments in the header are discarded in the process of reading the
 * image. Upon failure, this function returns 0, upon sucess it returns 1.
 ******************************************************************************/
int read_pgm_image(char *infilename, unsigned char **image, int *rows,
                   int *cols) {
    FILE *fp;
    char buf[71];

    /***************************************************************************
     * Open the input image file for reading if a filename was given. If no
     * filename was provided, set fp to read from standard input.
     ***************************************************************************/
    if (infilename == NULL)
        fp = stdin;
    else {
        if ((fp = fopen(infilename, "r")) == NULL) {
            fprintf(stderr, "Error reading the file %s in read_pgm_image().\n",
                    infilename);
            return (0);
        }
    }

    /***************************************************************************
     * Verify that the image is in PGM format, read in the number of columns
     * and rows in the image and scan past all of the header information.
     ***************************************************************************/
    fgets(buf, 70, fp);
    if (strncmp(buf, "P5", 2) != 0) {
        fprintf(stderr, "The file %s is not in PGM format in ", infilename);
        fprintf(stderr, "read_pgm_image().\n");
        if (fp != stdin)
            fclose(fp);
        return (0);
    }
    do {
        fgets(buf, 70, fp);
    } while (buf[0] == '#'); /* skip all comment lines */
    sscanf(buf, "%d %d", cols, rows);
    do {
        fgets(buf, 70, fp);
    } while (buf[0] == '#'); /* skip all comment lines */

    /***************************************************************************
     * Allocate memory to store the image then read the image from the file.
     ***************************************************************************/
    if (((*image) = (unsigned char *)malloc((*rows) * (*cols))) == NULL) {
        fprintf(stderr, "Memory allocation failure in read_pgm_image().\n");
        if (fp != stdin)
            fclose(fp);
        return (0);
    }
    if ((*rows) != fread((*image), (*cols), (*rows), fp)) {
        fprintf(stderr, "Error reading the image data in read_pgm_image().\n");
        if (fp != stdin)
            fclose(fp);
        free((*image));
        return (0);
    }

    if (fp != stdin)
        fclose(fp);
    return (1);
}

/******************************************************************************
 * Function: write_pgm_image
 * Purpose: This function writes an image in PGM format. The file is either
 * written to the file specified by outfilename or to standard output if
 * outfilename = NULL. A comment can be written to the header if coment != NULL.
 ******************************************************************************/
int write_pgm_image(char *outfilename, unsigned char *image, int rows,
                    int cols, char *comment, int maxval) {
    FILE *fp;

    /***************************************************************************
     * Open the output image file for writing if a filename was given. If no
     * filename was provided, set fp to write to standard output.
     ***************************************************************************/
    if (outfilename == NULL)
        fp = stdout;
    else {
        if ((fp = fopen(outfilename, "w")) == NULL) {
            fprintf(stderr, "Error writing the file %s in write_pgm_image().\n",
                    outfilename);
            return (0);
        }
    }

    /***************************************************************************
     * Write the header information to the PGM file.
     ***************************************************************************/
    fprintf(fp, "P5\n%d %d\n", cols, rows);
    if (comment != NULL)
        if (strlen(comment) <= 70)
            fprintf(fp, "# %s\n", comment);
    fprintf(fp, "%d\n", maxval);

    /***************************************************************************
     * Write the image data to the file.
     ***************************************************************************/
    if (rows != fwrite(image, cols, rows, fp)) {
        fprintf(stderr, "Error writing the image data in write_pgm_image().\n");
        if (fp != stdout)
            fclose(fp);
        return (0);
    }

    if (fp != stdout)
        fclose(fp);
    return (1);
}

/******************************************************************************
 * Function: read_ppm_image
 * Purpose: This function reads in an image in PPM format. The image can be
 * read in from either a file or from standard input. The image is only read
 * from standard input when infilename = NULL. Because the PPM format includes
 * the number of columns and the number of rows in the image, these are read
 * from the file. Memory to store the image is allocated in this function.
 * All comments in the header are discarded in the process of reading the
 * image. Upon failure, this function returns 0, upon sucess it returns 1.
 ******************************************************************************/
int read_ppm_image(char *infilename, unsigned char **image_red,
                   unsigned char **image_grn, unsigned char **image_blu, int *rows,
                   int *cols) {
    FILE *fp;
    char buf[71];
    int p, size;

    /***************************************************************************
     * Open the input image file for reading if a filename was given. If no
     * filename was provided, set fp to read from standard input.
     ***************************************************************************/
    if (infilename == NULL)
        fp = stdin;
    else {
        if ((fp = fopen(infilename, "r")) == NULL) {
            fprintf(stderr, "Error reading the file %s in read_ppm_image().\n",
                    infilename);
            return (0);
        }
    }

    /***************************************************************************
     * Verify that the image is in PPM format, read in the number of columns
     * and rows in the image and scan past all of the header information.
     ***************************************************************************/
    fgets(buf, 70, fp);
    if (strncmp(buf, "P6", 2) != 0) {
        fprintf(stderr, "The file %s is not in PPM format in ", infilename);
        fprintf(stderr, "read_ppm_image().\n");
        if (fp != stdin)
            fclose(fp);
        return (0);
    }
    do {
        fgets(buf, 70, fp);
    } while (buf[0] == '#'); /* skip all comment lines */
    sscanf(buf, "%d %d", cols, rows);
    do {
        fgets(buf, 70, fp);
    } while (buf[0] == '#'); /* skip all comment lines */

    /***************************************************************************
     * Allocate memory to store the image then read the image from the file.
     ***************************************************************************/
    if (((*image_red) = (unsigned char *)malloc((*rows) * (*cols))) == NULL) {
        fprintf(stderr, "Memory allocation failure in read_ppm_image().\n");
        if (fp != stdin)
            fclose(fp);
        return (0);
    }
    if (((*image_grn) = (unsigned char *)malloc((*rows) * (*cols))) == NULL) {
        fprintf(stderr, "Memory allocation failure in read_ppm_image().\n");
        if (fp != stdin)
            fclose(fp);
        return (0);
    }
    if (((*image_blu) = (unsigned char *)malloc((*rows) * (*cols))) == NULL) {
        fprintf(stderr, "Memory allocation failure in read_ppm_image().\n");
        if (fp != stdin)
            fclose(fp);
        return (0);
    }

    size = (*rows) * (*cols);
    for (p = 0; p < size; p++) {
        (*image_red)[p] = (unsigned char)fgetc(fp);
        (*image_grn)[p] = (unsigned char)fgetc(fp);
        (*image_blu)[p] = (unsigned char)fgetc(fp);
    }

    if (fp != stdin)
        fclose(fp);
    return (1);
}

/******************************************************************************
 * Function: write_ppm_image
 * Purpose: This function writes an image in PPM format. The file is either
 * written to the file specified by outfilename or to standard output if
 * outfilename = NULL. A comment can be written to the header if coment != NULL.
 ******************************************************************************/
int write_ppm_image(char *outfilename, unsigned char *image_red,
                    unsigned char *image_grn, unsigned char *image_blu, int rows,
                    int cols, char *comment, int maxval) {
    FILE *fp;
    long size, p;

    /***************************************************************************
     * Open the output image file for writing if a filename was given. If no
     * filename was provided, set fp to write to standard output.
     ***************************************************************************/
    if (outfilename == NULL)
        fp = stdout;
    else {
        if ((fp = fopen(outfilename, "w")) == NULL) {
            fprintf(stderr, "Error writing the file %s in write_pgm_image().\n",
                    outfilename);
            return (0);
        }
    }

    /***************************************************************************
     * Write the header information to the PGM file.
     ***************************************************************************/
    fprintf(fp, "P6\n%d %d\n", cols, rows);
    if (comment != NULL)
        if (strlen(comment) <= 70)
            fprintf(fp, "# %s\n", comment);
    fprintf(fp, "%d\n", maxval);

    /***************************************************************************
     * Write the image data to the file.
     ***************************************************************************/
    size = (long)rows * (long)cols;
    for (p = 0; p < size; p++) { /* Write the image in pixel interleaved format. */
        fputc(image_red[p], fp);
        fputc(image_grn[p], fp);
        fputc(image_blu[p], fp);
    }

    if (fp != stdout)
        fclose(fp);
    return (1);
}

/*******************************************************************************
 * --------------------------------------------
 *(c) 2001 University of South Florida, Tampa
 * Use, or copying without permission prohibited.
 * PERMISSION TO USE
 * In transmitting this software, permission to use for research and
 * educational purposes is hereby granted.  This software may be copied for
 * archival and backup purposes only.  This software may not be transmitted
 * to a third party without prior permission of the copyright holder. This
 * permission may be granted only by Mike Heath or Prof. Sudeep Sarkar of
 * University of South Florida (sarkar@csee.usf.edu). Acknowledgment as
 * appropriate is respectfully requested.
 *
 *  Heath, M., Sarkar, S., Sanocki, T., and Bowyer, K. Comparison of edge
 *    detectors: a methodology and initial study, Computer Vision and Image
 *    Understanding 69 (1), 38-54, January 1998.
 *  Heath, M., Sarkar, S., Sanocki, T. and Bowyer, K.W. A Robust Visual
 *    Method for Assessing the Relative Performance of Edge Detection
 *    Algorithms, IEEE Transactions on Pattern Analysis and Machine
 *    Intelligence 19 (12),  1338-1359, December 1997.
 *  ------------------------------------------------------
 *
 * PROGRAM: canny_edge
 * PURPOSE: This program implements a "Canny" edge detector. The processing
 * steps are as follows:
 *
 *   1) Convolve the image with a separable gaussian filter.
 *   2) Take the dx and dy the first derivatives using [-1,0,1] and [1,0,-1]'.
 *   3) Compute the magnitude: sqrt(dx*dx+dy*dy).
 *   4) Perform non-maximal suppression.
 *   5) Perform hysteresis.
 *
 * The user must input three parameters. These are as follows:
 *
 *   sigma = The standard deviation of the gaussian smoothing filter.
 *   tlow  = Specifies the low value to use in hysteresis. This is a
 *           fraction (0-1) of the computed high threshold edge strength value.
 *   thigh = Specifies the high value to use in hysteresis. This fraction (0-1)
 *           specifies the percentage point in a histogram of the gradient of
 *           the magnitude. Magnitude values of zero are not counted in the
 *           histogram.
 *
 * NAME: Mike Heath
 *       Computer Vision Laboratory
 *       University of South Floeida
 *       heath@csee.usf.edu
 *
 * DATE: 2/15/96
 *
 * Modified: 5/17/96 - To write out a floating point RAW headerless file of
 *                     the edge gradient "up the edge" where the angle is
 *                     defined in radians counterclockwise from the x direction.
 *                     (Mike Heath)
 *******************************************************************************/

typedef long long fixed;
#define fixeddot 16

#define VERBOSE 0
#define BOOSTBLURFACTOR 90.0

#include <math.h>
#include <stdio.h>
#include <stdlib.h>

int read_pgm_image(char *infilename, unsigned char **image, int *rows,
                   int *cols);
int write_pgm_image(char *outfilename, unsigned char *image, int rows,
                    int cols, char *comment, int maxval);

void canny(unsigned char *image, int rows, int cols, float sigma,
           float tlow, float thigh, unsigned char **edge, char *fname);
void gaussian_smooth(unsigned char *image, int rows, int cols, float sigma,
                     short int **smoothedim);
void make_gaussian_kernel(float sigma, float **kernel, int *windowsize);
void derrivative_x_y(short int *smoothedim, int rows, int cols,
                     short int **delta_x, short int **delta_y);
void magnitude_x_y(short int *delta_x, short int *delta_y, int rows, int cols,
                   short int **magnitude);
void apply_hysteresis(short int *mag, unsigned char *nms, int rows, int cols,
                      float tlow, float thigh, unsigned char *edge);
void radian_direction(short int *delta_x, short int *delta_y, int rows,
                      int cols, float **dir_radians, int xdirtag, int ydirtag);
double angle_radians(double x, double y);

int main(int argc, char *argv[]) {
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    char *infilename = NULL;  /* Name of the input image */
    char *dirfilename = NULL; /* Name of the output gradient direction image */
    char outfilename[128];    /* Name of the output "edge" image */
    char composedfname[128];  /* Name of the output "direction" image */
    unsigned char *image;     /* The input image */
    unsigned char *edge;      /* The output edge image */
    int rows, cols;           /* The dimensions of the image. */
    float sigma,              /* Standard deviation of the gaussian kernel. */
        tlow,                 /* Fraction of the high threshold in hysteresis. */
        thigh;                /* High hysteresis threshold control. The actual
                                 threshold is the (100 * thigh) percentage point
                                 in the histogram of the magnitude of the
                                 gradient image that passes non-maximal
                                 suppression. */

    /****************************************************************************
     * Get the command line arguments.
     ****************************************************************************/
    if (argc < 5) {
        fprintf(stderr, "\n<USAGE> %s image sigma tlow thigh [writedirim]\n", argv[0]);
        fprintf(stderr, "\n      image:      An image to process. Must be in ");
        fprintf(stderr, "PGM format.\n");
        fprintf(stderr, "      sigma:      Standard deviation of the gaussian");
        fprintf(stderr, " blur kernel.\n");
        fprintf(stderr, "      tlow:       Fraction (0.0-1.0) of the high ");
        fprintf(stderr, "edge strength threshold.\n");
        fprintf(stderr, "      thigh:      Fraction (0.0-1.0) of the distribution");
        fprintf(stderr, " of non-zero edge\n                  strengths for ");
        fprintf(stderr, "hysteresis. The fraction is used to compute\n");
        fprintf(stderr, "                  the high edge strength threshold.\n");
        fprintf(stderr, "      writedirim: Optional argument to output ");
        fprintf(stderr, "a floating point");
        fprintf(stderr, " direction image.\n\n");
        exit(1);
    }

    infilename = argv[1];
    sigma = atof(argv[2]);
    tlow = atof(argv[3]);
    thigh = atof(argv[4]);

    if (argc == 6)
        dirfilename = infilename;
    else
        dirfilename = NULL;

    /****************************************************************************
     * Read in the image. This read function allocates memory for the image.
     ****************************************************************************/
    if (VERBOSE)
        printf("Reading the image %s.\n", infilename);
    if (read_pgm_image(infilename, &image, &rows, &cols) == 0) {
        fprintf(stderr, "Error reading the input image, %s.\n", infilename);
        exit(1);
    }

    /****************************************************************************
     * Perform the edge detection. All of the work takes place here.
     ****************************************************************************/
    if (VERBOSE)
        printf("Starting Canny edge detection.\n");
    if (dirfilename != NULL) {
        sprintf(composedfname, "%s_s_%3.2f_l_%3.2f_h_%3.2f.fim", infilename,
                sigma, tlow, thigh);
        dirfilename = composedfname;
    }

    ///////
    canny(image, rows, cols, sigma, tlow, thigh, &edge, dirfilename);
    ///////

    /****************************************************************************
     * Write out the edge image to a file.
     ****************************************************************************/
    sprintf(outfilename, "%s_s_%3.2f_l_%3.2f_h_%3.2f.pgm", infilename,
            sigma, tlow, thigh);
    if (VERBOSE)
        printf("Writing the edge iname in the file %s.\n", outfilename);
    if (write_pgm_image(outfilename, edge, rows, cols, "", 255) == 0) {
        fprintf(stderr, "Error writing the edge image, %s.\n", outfilename);
        exit(1);
    }
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // Step 6: Calculate elapsed time
    float milliseconds = 0;
    hipEventElapsedTime(&milliseconds, start, stop);

    // Print the time taken
    printf("Time elapsed: %f ms", milliseconds);

    // Step 7: Destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);
    return 0;
}

/*******************************************************************************
 * PROCEDURE: canny
 * PURPOSE: To perform canny edge detection.
 * NAME: Mike Heath
 * DATE: 2/15/96
 *******************************************************************************/
void canny(unsigned char *image, int rows, int cols, float sigma,
           float tlow, float thigh, unsigned char **edge, char *fname) {
    FILE *fpdir = NULL;    /* File to write the gradient image to.     */
    unsigned char *nms;    /* Points that are local maximal magnitude. */
    short int *smoothedim, /* The image after gaussian smoothing.      */
        *delta_x,          /* The first devivative image, x-direction. */
        *delta_y,          /* The first derivative image, y-direction. */
        *magnitude;        /* The magnitude of the gadient image.      */
    int r, c, pos;
    float *dir_radians = NULL; /* Gradient direction image.                */

    /****************************************************************************
     * Perform gaussian smoothing on the image using the input standard
     * deviation.
     ****************************************************************************/
    if (VERBOSE)
        printf("Smoothing the image using a gaussian kernel.\n");
    gaussian_smooth(image, rows, cols, sigma, &smoothedim);

    /****************************************************************************
     * Compute the first derivative in the x and y directions.
     ****************************************************************************/
    if (VERBOSE)
        printf("Computing the X and Y first derivatives.\n");
    derrivative_x_y(smoothedim, rows, cols, &delta_x, &delta_y);

    /****************************************************************************
     * This option to write out the direction of the edge gradient was added
     * to make the information available for computing an edge quality figure
     * of merit.
     ****************************************************************************/
    if (fname != NULL) {
        /*************************************************************************
         * Compute the direction up the gradient, in radians that are
         * specified counteclockwise from the positive x-axis.
         *************************************************************************/
        radian_direction(delta_x, delta_y, rows, cols, &dir_radians, -1, -1);

        /*************************************************************************
         * Write the gradient direction image out to a file.
         *************************************************************************/
        if ((fpdir = fopen(fname, "wb")) == NULL) {
            fprintf(stderr, "Error opening the file %s for writing.\n", fname);
            exit(1);
        }
        fwrite(dir_radians, sizeof(float), rows * cols, fpdir);
        fclose(fpdir);
        free(dir_radians);
    }

    /****************************************************************************
     * Compute the magnitude of the gradient.
     ****************************************************************************/
    if (VERBOSE)
        printf("Computing the magnitude of the gradient.\n");
    magnitude_x_y(delta_x, delta_y, rows, cols, &magnitude);

    /****************************************************************************
     * Perform non-maximal suppression.
     ****************************************************************************/
    if (VERBOSE)
        printf("Doing the non-maximal suppression.\n");
    if ((nms = (unsigned char *)malloc(rows * cols * sizeof(unsigned char))) == NULL) {
        fprintf(stderr, "Error allocating the nms image.\n");
        exit(1);
    }
    non_max_supp(magnitude, delta_x, delta_y, rows, cols, nms);

    /****************************************************************************
     * Use hysteresis to mark the edge pixels.
     ****************************************************************************/
    if (VERBOSE)
        printf("Doing hysteresis thresholding.\n");
    if ((*edge = (unsigned char *)malloc(rows * cols * sizeof(unsigned char))) == NULL) {
        fprintf(stderr, "Error allocating the edge image.\n");
        exit(1);
    }
    apply_hysteresis(magnitude, nms, rows, cols, tlow, thigh, *edge);

    /****************************************************************************
     * Free all of the memory that we allocated except for the edge image that
     * is still being used to store out result.
     ****************************************************************************/
    free(smoothedim);
    free(delta_x);
    free(delta_y);
    free(magnitude);
    free(nms);
}

/*******************************************************************************
 * Procedure: radian_direction
 * Purpose: To compute a direction of the gradient image from component dx and
 * dy images. Because not all derriviatives are computed in the same way, this
 * code allows for dx or dy to have been calculated in different ways.
 *
 * FOR X:  xdirtag = -1  for  [-1 0  1]
 *         xdirtag =  1  for  [ 1 0 -1]
 *
 * FOR Y:  ydirtag = -1  for  [-1 0  1]'
 *         ydirtag =  1  for  [ 1 0 -1]'
 *
 * The resulting angle is in radians measured counterclockwise from the
 * xdirection. The angle points "up the gradient".
 *******************************************************************************/
void radian_direction(short int *delta_x, short int *delta_y, int rows,
                      int cols, float **dir_radians, int xdirtag, int ydirtag) {
    int r, c, pos;
    float *dirim = NULL;
    double dx, dy;

    /****************************************************************************
     * Allocate an image to store the direction of the gradient.
     ****************************************************************************/
    if ((dirim = (float *)malloc(rows * cols * sizeof(float))) == NULL) {
        fprintf(stderr, "Error allocating the gradient direction image.\n");
        exit(1);
    }
    *dir_radians = dirim;

    for (r = 0, pos = 0; r < rows; r++) {
        for (c = 0; c < cols; c++, pos++) {
            dx = (double)delta_x[pos];
            dy = (double)delta_y[pos];

            if (xdirtag == 1)
                dx = -dx;
            if (ydirtag == -1)
                dy = -dy;

            dirim[pos] = (float)angle_radians(dx, dy);
        }
    }
}

/*******************************************************************************
 * FUNCTION: angle_radians
 * PURPOSE: This procedure computes the angle of a vector with components x and
 * y. It returns this angle in radians with the answer being in the range
 * 0 <= angle <2*PI.
 *******************************************************************************/
double angle_radians(double x, double y) {
    double xu, yu, ang;

    xu = fabs(x);
    yu = fabs(y);

    if ((xu == 0) && (yu == 0))
        return (0);

    ang = atan(yu / xu);

    if (x >= 0) {
        if (y >= 0)
            return (ang);
        else
            return (2 * M_PI - ang);
    } else {
        if (y >= 0)
            return (M_PI - ang);
        else
            return (M_PI + ang);
    }
}

/*******************************************************************************
 * PROCEDURE: magnitude_x_y
 * PURPOSE: Compute the magnitude of the gradient. This is the square root of
 * the sum of the squared derivative values.
 * NAME: Mike Heath
 * DATE: 2/15/96
 *******************************************************************************/

__global__ void magnitude_kernal(short int *magnitudeDevice, short int *delta_xDevice, short int *delta_yDevice, int rows, int cols) {

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y;

    int pos = r * cols + c;
    int sq1, sq2;

    if (r < rows && c < cols) {
        sq1 = (int)delta_xDevice[pos] * (int)delta_xDevice[pos];
        sq2 = (int)delta_yDevice[pos] * (int)delta_yDevice[pos];
        magnitudeDevice[pos] = (short)(0.5 + sqrt((float)sq1 + (float)sq2));
    }
}

void magnitude_x_y(short int *delta_x, short int *delta_y, int rows, int cols,
                   short int **magnitude) {
    int r, c, pos, sq1, sq2;

    /****************************************************************************
     * Allocate an image to store the magnitude of the gradient.
     ****************************************************************************/
    short int *magnitudeDevice;
    short int *delta_xDevice;
    short int *delta_yDevice;

    if ((*magnitude = (short *)malloc(rows * cols * sizeof(short))) == NULL) {
        fprintf(stderr, "Error allocating the magnitude image.\n");
        exit(1);
    }

    if ((hipMalloc((void **)&magnitudeDevice, rows * cols * sizeof(short))) != hipSuccess) {
        fprintf(stderr, "Error allocating the magnitudeDevice image.\n");
        exit(1);
    }

    if ((hipMalloc((void **)&delta_xDevice, rows * cols * sizeof(short))) != hipSuccess) {
        fprintf(stderr, "Error allocating the delta_xDevice image.\n");
        exit(1);
    }
    if ((hipMalloc((void **)&delta_yDevice, rows * cols * sizeof(short))) != hipSuccess) {
        fprintf(stderr, "Error allocating the delta_yDevice image.\n");
        exit(1);
    }

    hipError_t err = hipMemcpy(delta_yDevice, delta_y, rows * cols * sizeof(short int), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying delta_y from device to host: %s\n", hipGetErrorString(err));
        exit(1);
    }

    if (hipMemcpy(delta_xDevice, delta_x, rows * cols * sizeof(short int), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Error copying delta_x from device to host.\n");
        exit(1);
    }

    dim3 block(256);
    dim3 grid((cols + 255) / 256, rows);
    magnitude_kernal<<<grid, block>>>(magnitudeDevice, delta_xDevice, delta_yDevice, rows, cols);
    hipDeviceSynchronize();

    if (hipMemcpy(*magnitude, magnitudeDevice, rows * cols * sizeof(short int), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "Error copying delta_x from device to host.\n");
        exit(1);
    }

    // for (r = 0, pos = 0; r < rows; r++) {
    //     for (c = 0; c < cols; c++, pos++) {
    //         sq1 = (int)delta_x[pos] * (int)delta_x[pos];
    //         sq2 = (int)delta_y[pos] * (int)delta_y[pos];
    //         (*magnitude)[pos] = (short)(0.5 + sqrt((float)sq1 + (float)sq2));
    //     }
    // }
}
/*******************************************************************************
 * PROCEDURE: derrivative_x_y
 * PURPOSE: Compute the first derivative of the image in both the x any y
 * directions. The differential filters that are used are:
 *
 *                                          -1
 *         dx =  -1 0 +1     and       dy =  0
 *                                          +1
 *
 * NAME: Mike Heath
 * DATE: 2/15/96
 *******************************************************************************/

__global__ void derivateX(short int *delta_yDevice, short int *smoothedim, int rows, int cols) {
    int tid = blockIdx.x;

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y;

    if (r < rows && c < cols) {
        int pos = r * cols + c;
        if (r > 0 && r < cols - 1) {
            delta_yDevice[pos] = smoothedim[pos + cols] - smoothedim[pos - cols];
        } else if (r == 0) {
            delta_yDevice[pos] = smoothedim[pos + cols] - smoothedim[pos];
        } else if (r == rows - 1) {
            delta_yDevice[pos] = smoothedim[pos] - smoothedim[pos - cols];
        }
    }
}

__global__ void derivateY(short int *delta_xDevice, short int *smoothedim, int rows, int cols) {
    int tid = blockIdx.x;

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y;
    if (r < rows && c < cols) {
        int pos = r * cols + c;
        if (c > 0 && c < cols - 1) {
            delta_xDevice[pos] = smoothedim[pos + 1] - smoothedim[pos - 1];
        } else if (c == 0) {
            delta_xDevice[pos] = smoothedim[pos + 1] - smoothedim[pos];
        } else {
            delta_xDevice[pos] = smoothedim[pos] - smoothedim[pos - 1];
        }
    }
}

void derrivative_x_y(short int *smoothedim, int rows, int cols,
                     short int **delta_x, short int **delta_y) {
    /****************************************************************************
     * Allocate images to store the derivatives.
     ****************************************************************************/
    short int *delta_xDevice;
    short int *delta_yDevice;
    short int *smoothedimDevice;

    if (((*delta_x) = (short *)malloc(rows * cols * sizeof(short))) == NULL) {
        fprintf(stderr, "Error allocating the delta_x image.\n");
        exit(1);
    }
    if (((*delta_y) = (short *)malloc(rows * cols * sizeof(short))) == NULL) {
        fprintf(stderr, "Error allocating the delta_x image.\n");
        exit(1);
    }

    if ((hipMalloc((void **)&delta_xDevice, rows * cols * sizeof(short))) != hipSuccess) {
        fprintf(stderr, "Error allocating the delta_xDevice image.\n");
        exit(1);
    }
    if ((hipMalloc((void **)&delta_yDevice, rows * cols * sizeof(short))) != hipSuccess) {
        fprintf(stderr, "Error allocating the delta_yDevice image.\n");
        exit(1);
    }

    if (hipMalloc((void **)&smoothedimDevice, rows * cols * sizeof(short int)) != hipSuccess) {
        fprintf(stderr, "Error allocating smoothedimDevice on GPU.\n");
        exit(1);
    }

    if (hipMemcpy(smoothedimDevice, smoothedim, rows * cols * sizeof(short int), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Error copying smoothedim from host to device.\n");
        exit(1);
    }

    dim3 block(256);
    dim3 grid((cols + 255) / 256, rows);
    derivateX<<<grid, block>>>(delta_yDevice, smoothedimDevice, rows, cols);
    derivateY<<<grid, block>>>(delta_xDevice, smoothedimDevice, rows, cols);
    hipDeviceSynchronize();
    hipError_t err = hipMemcpy(*delta_y, delta_yDevice, rows * cols * sizeof(short int), hipMemcpyDeviceToHost);
    if (err != hipSuccess) {
        fprintf(stderr, "Error copying delta_y from device to host: %s\n", hipGetErrorString(err));
        exit(1);
    }

    if (hipMemcpy(*delta_x, delta_xDevice, rows * cols * sizeof(short int), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "Error copying delta_x from device to host.\n");
        exit(1);
    }
}

/*******************************************************************************
 * PROCEDURE: gaussian_smooth
 * PURPOSE: Blur an image with a gaussian filter.
 * NAME: Mike Heath
 * DATE: 2/15/96
 *******************************************************************************/

__global__ void blurx(float *tempim, unsigned char *imageDevice, float *kernalDevice, int center, int rows, int cols) {
    float dot = 0.0;
    float sum = 0.0;

    int tid = blockIdx.x;

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y;

    if (r < rows && c < cols) {
        for (int cc = (-center); cc <= center; cc++) {
            if (((c + cc) >= 0) && ((c + cc) < cols)) {
                dot += (float)imageDevice[r * cols + (c + cc)] * kernalDevice[center + cc];
                sum += kernalDevice[center + cc];
            }
        }
        tempim[r * cols + c] = dot / sum;
    }
}

const int BLOCKSIZE = 256;
const int MAX_KERNEL_SIZE = 30; 
__constant__ float kernelDeviceConstant[MAX_KERNEL_SIZE];  
__global__ void blurxShared(float *tempim, unsigned char *imageDevice, float *kernalDevice, int center, int rows, int cols) {
    float dot = 0.0;
    float sum = 0.0;

    extern __shared__ unsigned char imageShared[];
    int tid = blockIdx.x;
    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y;

    int idx = r * cols + c;
    int localIdx = threadIdx.x + center;
    if (c < cols) {
        imageShared[localIdx] = imageDevice[idx]; 
    } else {
        imageShared[localIdx] = 0; 
    }

    __syncthreads();
    if (r < rows && c < cols) {
        for (int cc = (-center); cc <= center; cc++) {
            dot += (float)imageShared[localIdx + cc] * kernelDeviceConstant[center + cc];
            sum += kernelDeviceConstant[center + cc];
        }
        tempim[idx] = dot / sum;
    }
}

// Revised vertical blur kernel using shared memory
__global__ void bluryShared(float *tempim, short int *smoothedim, float *kernalDevice, 
    int center, int rows, int cols) {
    float dot = 0.0f;
    float sum = 0.0f;

    extern __shared__ float tempimShared[];

    int c = blockIdx.x;  
    int r = blockIdx.y * blockDim.y + threadIdx.y;  // global row index

    int localIdx = threadIdx.y + center;
    if (r < rows && c < cols)
    tempimShared[localIdx] = tempim[r * cols + c];
    else
    tempimShared[localIdx] = 0.0f;

    __syncthreads();

    if (r < rows && c < cols) {
    for (int rr = -center; rr <= center; rr++) {
        dot += tempimShared[localIdx + rr] * kernelDeviceConstant[center + rr];
        sum += kernelDeviceConstant[center + rr];
    }
    smoothedim[r * cols + c] = (short int)(dot * BOOSTBLURFACTOR / sum + 0.5f);
    }
}


__global__ void blurY(float *tempim, short int *smoothedim, float *kernalDevice, int center, int rows, int cols) {
    float dot = 0.0;
    float sum = 0.0;

    int tid = blockIdx.x;

    int c = blockIdx.x * blockDim.x + threadIdx.x;
    int r = blockIdx.y;

    if (r < rows && c < cols) {
        for (int rr = (-center); rr <= center; rr++) {
            if (((r + rr) >= 0) && ((r + rr) < rows)) {
                dot += tempim[(r + rr) * cols + c] * kernalDevice[center + rr];
                sum += kernalDevice[center + rr];
            }
        }
        (smoothedim)[r * cols + c] = (short int)(dot * BOOSTBLURFACTOR / sum + 0.5);
    }
}

void gaussian_smooth(unsigned char *image, int rows, int cols, float sigma,
                     short int **smoothedim) {
    int windowsize, /* Dimension of the gaussian kernel. */
        center;     /* Half of the windowsize. */
    float *tempim,  /* Buffer for separable filter gaussian smoothing. */
        *kernel,    /* A one dimensional gaussian kernel. */
        dot;        /* Dot product summing variable. */

    float *kernelDevice;
    short int *smoothedimDevice;
    unsigned char *imageDevice;
    /****************************************************************************
     * Create a 1-dimensional gaussian smoothing kernel.
     ****************************************************************************/
    if (VERBOSE)
        printf("   Computing the gaussian smoothing kernel.\n");
    make_gaussian_kernel(sigma, &kernel, &windowsize);
    center = windowsize / 2;

    /****************************************************************************
     * Allocate a temporary buffer image and the smoothed image.
     ****************************************************************************/
    if (hipMalloc((void **)&tempim, rows * cols * sizeof(float)) != hipSuccess) {
        fprintf(stderr, "Error allocating the buffer image on GPU.\n");
        exit(1);
    }

    if (((*smoothedim) = (short int *)malloc(rows * cols * sizeof(short int))) == NULL) {
        fprintf(stderr, "Error allocating the smoothed image.\n");
        exit(1);
    }

    if (hipMalloc((void **)&smoothedimDevice, rows * cols * sizeof(short int)) != hipSuccess) {
        fprintf(stderr, "Error allocating smoothedimDevice GPU.\n");
        exit(1);
    }

    if (hipMalloc((void **)&kernelDevice, (windowsize) * sizeof(float)) != hipSuccess) {
        fprintf(stderr, "Error allocating the gaussian kernel array on GPU.\n");
        exit(1);
    }

    if (hipMalloc((void **)&imageDevice, (rows) * (cols) * sizeof(unsigned char)) != hipSuccess) {
        fprintf(stderr, "Memory allocation failure on GPU in n");
        exit(0);
    }

    if (hipMemcpy(kernelDevice, kernel, windowsize * sizeof(float), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Error copying the gaussian kernel array from host to device.\n");
        exit(1);
    }

    if (hipMemcpy(imageDevice, image, rows * cols * sizeof(unsigned char), hipMemcpyHostToDevice) != hipSuccess) {
        fprintf(stderr, "Error copying the image from host to device.\n");
        exit(1);
    }


    if (windowsize > MAX_KERNEL_SIZE) {
        fprintf(stderr, "Error: Kernel size exceeds the maximum constant memory size.\n");
        exit(1);
    }

    // Copy kernel from host to constant memory
    hipError_t status = hipMemcpyToSymbol(HIP_SYMBOL(kernelDeviceConstant), kernel, windowsize * sizeof(float));
    if (status != hipSuccess) {
        fprintf(stderr, "Error copying the Gaussian kernel to constant memory: %s\n", hipGetErrorString(status));
        exit(1);
    }
    printf("Bluring the image in the X-direction.\n");
    dim3 block(256);
    dim3 grid((cols + 255) / 256, rows);
    blurxShared<<<grid, block, BLOCKSIZE + 2 * center>>>(tempim, imageDevice, kernelDevice, center, rows, cols);
    // blurx<<<grid, block>>>(tempim, imageDevice, kernelDevice, center, rows, cols);
    hipDeviceSynchronize();

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
        exit(1);
    }
    printf("Bluring the image in the Y-direction.\n");

    // blurY<<<grid, block>>>(tempim, smoothedimDevice, kernelDevice, center, rows, cols);

    dim3 block1(1, BLOCKSIZE);
    dim3 grid1(cols, (rows + BLOCKSIZE - 1) / BLOCKSIZE);
    size_t sharedSize = (BLOCKSIZE + 2 * center) * sizeof(float);
    bluryShared<<<grid1, block1, sharedSize>>>(tempim, smoothedimDevice, kernelDevice, center, rows, cols);


    err = hipGetLastError();
    if (err != hipSuccess) {
        fprintf(stderr, "Kernel launch error: %s\n", hipGetErrorString(err));
        exit(1);
    }
    hipDeviceSynchronize();
    if (hipMemcpy(*smoothedim, smoothedimDevice, rows * cols * sizeof(short int), hipMemcpyDeviceToHost) != hipSuccess) {
        fprintf(stderr, "Error copying the gaussian kernel array from host to device.\n");
        exit(1);
    }

    hipFree(kernelDevice);
    hipFree(imageDevice);
    hipFree(smoothedimDevice);
    hipFree(tempim);
    free(kernel);
}

/*******************************************************************************
 * PROCEDURE: make_gaussian_kernel
 * PURPOSE: Create a one dimensional gaussian kernel.
 * NAME: Mike Heath
 * DATE: 2/15/96
 *******************************************************************************/
void make_gaussian_kernel(float sigma, float **kernel, int *windowsize) {
    int i, center;
    float x, fx, sum = 0.0;

    *windowsize = 1 + 2 * ceil(2.5 * sigma);
    center = (*windowsize) / 2;

    if (VERBOSE)
        printf("      The kernel has %d elements.\n", *windowsize);
    if ((*kernel = (float *)malloc((*windowsize) * sizeof(float))) == NULL) {
        fprintf(stderr, "Error callocing the gaussian kernel array.\n");
        exit(1);
    }

    for (i = 0; i < (*windowsize); i++) {
        x = (float)(i - center);
        fx = pow(2.71828, -0.5 * x * x / (sigma * sigma)) / (sigma * sqrt(6.2831853));
        (*kernel)[i] = fx;
        sum += fx;
    }

    for (i = 0; i < (*windowsize); i++)
        (*kernel)[i] /= sum;

    if (VERBOSE) {
        printf("The filter coefficients are:\n");
        for (i = 0; i < (*windowsize); i++)
            printf("kernel[%d] = %f\n", i, (*kernel)[i]);
    }
}
